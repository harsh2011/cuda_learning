#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 3

__global__ void matrix_mul(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int sum = 0;
    // printf("## i %d j %d ", i, j);
    if (i < n && j < n) {
        for (int k = 0; k < n; k++){
            sum += a[i * n + k] * b[k * n + j];
            printf("i%d j%d # %d + %d = %d \n", i, j, a[i * n + k] , b[k * n + j] , sum);
        }
        c[i * n + j] = sum;
    }
}

int main() {
    int n = N;
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = n * n * sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++) {
            a[i * n + j] = i + j;
            b[i * n + j] = i * j;
        }

    printf("array:a \n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++)
            printf("%d ", a[i * n + j]);
        printf("\n");
    }
    printf("array:b \n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++)
            printf("%d ", b[i * n + j]);
        printf("\n");
    }
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 blockSize(N, N);
    dim3 gridSize((n + N - 1) / N, (n + N - 1) / N);
    matrix_mul<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    printf("array:c \n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++)
            printf("%d ", c[i * n + j]);
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}
