#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <Python.h>
#include <hip/hip_runtime.h>
#include <numpy/arrayobject.h>

#define NPY_NO_DEPRECATED_API NPY_1_7_API_VERSION


struct Weights {
    float* matrix;
    int ndims;
    int *shape;
    long int size;
};

struct Inputs {
    float* matrix;
    int ndims;
    int *shape;
    long int size;
};


PyObject* read_numpy_file(const char* file_path) {
    // Import the Python module containing the function
    PyObject* numpy_module = PyImport_ImportModule("numpy");
    if (numpy_module == nullptr) {
        PyErr_Print();
        return nullptr;
    }

    // Get the reference to the function
    PyObject* numpy_function = PyObject_GetAttrString(numpy_module, "load");
    if (numpy_function == nullptr) {
        PyErr_Print();
        Py_DECREF(numpy_module);
        return nullptr;
    }

    // Create the arguments tuple
    PyObject* args = PyTuple_New(1);
    PyTuple_SetItem(args, 0, PyUnicode_FromString(file_path));

    // Call the Python function with the arguments
    PyObject* result = PyObject_CallObject(numpy_function, args);
    if (result == nullptr) {
        PyErr_Print();
        Py_DECREF(numpy_module);
        Py_DECREF(numpy_function);
        Py_DECREF(args);
        return nullptr;
    }

    // Print the shape of the NumPy array
    PyObject* shape = PyObject_GetAttrString(result, "shape");
    if (shape != nullptr) {
        PyObject* repr = PyObject_Repr(shape);
        const char* str = PyUnicode_AsUTF8(repr);
        printf("Shape: %s\n", str);
        Py_DECREF(repr);
        Py_DECREF(shape);
    } else {
        printf("Failed to get shape.\n");
    }

    // Clean up references
    Py_DECREF(numpy_module);
    Py_DECREF(numpy_function);
    Py_DECREF(args);
    return result;
}

PyArrayObject* read_weights_from_numpy(const char* file_path, int print) {
    // Call the Python function and get the PyObject reference to the NumPy array
    PyObject* numpy_array = read_numpy_file(file_path);
    if (numpy_array == nullptr) {
        // Handle error
        return nullptr;
    }

    // Use the PyObject reference to the NumPy array as needed
    // Example: Print the array
    if (print == 1){
        PyObject* repr = PyObject_Repr(numpy_array);
        const char* str = PyUnicode_AsUTF8(repr);
        printf("%s\n", str);
    }

    // Convert the result to a NumPy array
    PyArrayObject* array = reinterpret_cast<PyArrayObject*>(numpy_array);

    return array;
}

int get_numpy_ndims(PyArrayObject* array){
    // Get number of dimensions
    int ndim = PyArray_NDIM(array);
    printf("%d \n", ndim);
    return ndim;
}


long int get_numpy_size(PyArrayObject* array){
    // Get the total size of the array
    npy_intp total_size_intp = PyArray_SIZE(array);
    printf("%" NPY_INTP_FMT "\n", total_size_intp);
    
    long int total_size = static_cast<long int>(total_size_intp);
    return total_size;
}



float* convert_PyArrayObject_to_float(PyArrayObject* array, int print, int *shape, int ndim) {

    printf("values %d %d \n", PyArray_TYPE(array), NPY_DOUBLE);
    // Check the data type of the numpy array
    if (PyArray_TYPE(array) != NPY_FLOAT32) {
        printf("Input numpy array is not of type float.\n");
    }

    // Convert the weight into float
    float* matrix = static_cast<float*>(PyArray_DATA(array));

    // Printing weights for checking it after conversion 
    if (print == 1){
        if (ndim == 2){
            for (int i = 0; i < shape[0]; ++i) {
                if (i ==0){
                    for (int j = 0; j < shape[1]; ++j) {
                        // matrix[j+(i*shape[1])] = 0.5f;
                        if(j<3 || j>shape[1]-3){
                            printf("%d index %.5f ", j+(i*shape[1]), matrix[j+(i*shape[1])]);
                        }
                    }
                    printf("\n");
                }
            }
        }
        else if (ndim == 1){
            for (int i = 0; i < shape[0]; ++i) {
                // matrix[i] = 0.5f;
                if(i<5 || i>shape[0]-5){
                    printf("%d index %.5f ", i, matrix[i]);
                }
                
            }
            printf("\n");
        }
       
    }

    return matrix; 
}

float* move_weight_to_cuda(float* weights ,long int total_size){
    // Allocate CUDA device memory
    float* d_data;
    printf("total size %d \n", total_size);

    hipError_t cudaStatus;
    cudaStatus = hipMalloc((void**)&d_data, total_size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        // Handle the error or return an error code
    }
    // Copy the array data from host (CPU) to device (CUDA)
    cudaStatus = hipMemcpy(d_data, weights, total_size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        // Handle the error or return an error code
    }
    return d_data;
}

void get_numpy_shape(PyArrayObject* array, Weights& weights, int ndim){
    // Get the dimensions of the array
    npy_intp* shape = PyArray_DIMS(array);
    weights.shape = (int *)malloc(ndim * sizeof(int));
    for (int i = 0; i < ndim; i++) {
        weights.shape[i] = static_cast<int>(shape[i]);
    }
}

Weights read_weights(const char* file_path, int print){
    Weights weight;

    // reading numpy weights
    PyArrayObject* array = read_weights_from_numpy(file_path, print);
    if (array == nullptr) {
        // Handle error
        weight.matrix = nullptr;
        return weight;
    }

    printf("111.111\n");

    int ndims = get_numpy_ndims(array);
    get_numpy_shape(array, weight, ndims);

    printf("111.222\n");
    
    long int size = get_numpy_size(array);
    float* matrix = convert_PyArrayObject_to_float(array, print, weight.shape, ndims);
    
    printf("111.333\n");
    // Release the PyObject reference
    

    float* cuda_weights = move_weight_to_cuda(matrix, size);
    
    printf("111.444\n");

    printf("################################ \n");
    Py_DECREF(array);




    weight.ndims = ndims;
    weight.size = size;
    weight.matrix = cuda_weights;

    return weight;
}

void get_numpy_shape(PyArrayObject* array, Inputs& weights, int ndim){
    // Get the dimensions of the array
    npy_intp* shape = PyArray_DIMS(array);
    weights.shape = (int *)malloc(ndim * sizeof(int));
    for (int i = 0; i < ndim; i++) {
        weights.shape[i] = static_cast<int>(shape[i]);
    }
}

Inputs read_image(const char* file_path, int print){
    Inputs input;

    PyArrayObject* array = read_weights_from_numpy(file_path, print);
    if (array == nullptr) {
        // Handle error
        input.matrix = nullptr;
        return input;
    }

    int ndims = get_numpy_ndims(array);
    get_numpy_shape(array, input, ndims);
    
    long int size = get_numpy_size(array);
    float* images = convert_PyArrayObject_to_float(array, print, input.shape, ndims);
    
    // Release the PyObject reference
    float* cuda_images = move_weight_to_cuda(images, size);

    Py_DECREF(array);


    input.ndims = ndims;
    input.size = size;
    input.matrix = cuda_images;

    return input;
}

__global__ void matrixMulKernel(float* matrixA, float* matrixB, float* matrixC, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        float sum = 0.0f;
        for (int k = 0; k < colsA; k++) {
            sum += matrixA[row * colsA + k] * matrixB[k * colsB + col];
        }
        matrixC[row * colsB + col] = sum;

        // Print intermediate result
        // printf("Intermediate result at rowsA %d, colsB %d  [%d][%d]: %.5f\n", rowsA, colsB, row, col, sum);
    }
}


__global__ void matrixAddKernel(float* matrixA, float* matrixB, float* matrixC, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        matrixC[index] = matrixA[index] + matrixB[index];
    }
}

__global__ void softmaxKernel(float* input, float* output, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;

        // Compute the exponential of each element
        float expVal = expf(input[index]);

        // Compute the sum of exponentials for the row
        float sumExp = 0.0f;
        for (int i = 0; i < cols; ++i) {
            sumExp += expf(input[row * cols + i]);
        }

        // Compute the softmax value for the element
        output[index] = expVal / sumExp;
    }
}



float* matrixMul(float* matrixA, float* matrixB, int rowsA, int colsA, int rowsB, int colsB){
    float* matrixC;

    printf(" ROW A %d  COLS A %d \n",rowsA, colsA);
    printf(" ROW B %d  COLS B %d \n",rowsB, colsB);

    hipMalloc((void **)&matrixC, rowsA * colsB * sizeof(float));

    dim3 blockSize(16, 16);
    dim3 gridSize((colsB + blockSize.x - 1) / blockSize.x, (rowsA + blockSize.y - 1) / blockSize.y);

    matrixMulKernel<<<gridSize, blockSize>>>(matrixA, matrixB, matrixC, rowsA, colsA, colsB);

    return matrixC;
}

float* matrixAdd(float* matrixA, float* matrixB, int rows, int cols){
    float* matrixC;

    printf(" ROW %d  COLS %d \n",rows, cols);

    hipMalloc((void **)&matrixC, rows * cols * sizeof(float));

    dim3 blockSize(16, 16);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);

    matrixAddKernel<<<gridSize, blockSize>>>(matrixA, matrixB, matrixC, rows, cols);

    return matrixC;
}


float* softmax(float* input, int rows, int cols){
    float* matrixC;

    printf(" ROW %d  COLS %d \n",rows, cols);

    hipMalloc((void **)&matrixC, rows * cols * sizeof(float));

    dim3 blockSize(16, 16);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);

    softmaxKernel<<<gridSize, blockSize>>>(input, matrixC, rows, cols);

    return matrixC;
}




int main() {
    // Initialize the Python interpreter
    Py_Initialize();

    // Ensure that NumPy is available
    import_array();
    Weights fc1_w = read_weights("./mlp/mnist_mlp/model/fc1.weight.npy", 0);
    Weights fc2_w = read_weights("./mlp/mnist_mlp/model/fc2.weight.npy", 0);
    Weights fc3_w = read_weights("./mlp/mnist_mlp/model/fc3.weight.npy", 0);
    printf("#############################\n");
    Weights fc1_b = read_weights("./mlp/mnist_mlp/model/fc1.bias.npy", 0);
    Weights fc2_b = read_weights("./mlp/mnist_mlp/model/fc2.bias.npy", 0);
    Weights fc3_b = read_weights("./mlp/mnist_mlp/model/fc3.bias.npy", 0);

    //Read image
    Inputs image = read_image("./mlp/mnist_mlp/images/1.npy", 0);
    // Inputs image = read_image("./mlp/mnist_mlp/2.npy", 0);

    // Finalize the Python interpreter
    Py_Finalize();

    int output_row = 1;
    int output_col = 10;

    float* matrixC = nullptr;
    
    matrixC = matrixMul(image.matrix, fc1_w.matrix, image.shape[0], image.shape[1], fc1_w.shape[0], fc1_w.shape[1]);
    matrixC = matrixAdd(matrixC, fc1_b.matrix, fc1_b.shape[0], fc1_b.shape[1]);

    matrixC = matrixMul(matrixC, fc2_w.matrix, fc1_b.shape[0], fc1_b.shape[1], fc2_w.shape[0], fc2_w.shape[1]);
    matrixC = matrixAdd(matrixC, fc2_b.matrix, fc2_b.shape[0], fc2_b.shape[1]);

    matrixC = matrixMul(matrixC, fc3_w.matrix, fc2_b.shape[0], fc2_b.shape[1], fc3_w.shape[0], fc3_w.shape[1]);
    matrixC = matrixAdd(matrixC, fc3_b.matrix, fc3_b.shape[0], fc3_b.shape[1]);

    matrixC = softmax(matrixC, fc3_b.shape[0], fc3_b.shape[1]);

    printf("output shape 1: %d, %d", fc3_b.shape[0], fc3_b.shape[1]);

    float* C = (float *)malloc(output_row * output_col * sizeof(float));

    hipMemcpy(C, matrixC,  output_row * output_col * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    printf("rowsA %d\n", output_row);
    printf("colsB %d\n", output_col);
    for (int i = 0; i < output_row; i++) {
        for (int j = 0; j < output_col; j++){
            printf("%f ", C[i * output_row + j]);
            
        }
        printf("\n");
    }


    sleep(5);
 
    // Clean up CUDA device memory
    hipFree(fc1_w.matrix);
    hipFree(fc2_w.matrix);
    hipFree(fc3_w.matrix);
    
    hipFree(fc1_b.matrix);
    hipFree(fc2_b.matrix);
    hipFree(fc3_b.matrix);

    return 0;
}